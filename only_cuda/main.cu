﻿#include <iostream>
#include <vector>
// #include <filesystem>
#include <opencv2/opencv.hpp>

#include "hip/hip_runtime.h"
#include ""
#include "cv_kernels.cuh"
#include "../cv_methods.hpp"

#define CUDA_BLOCK_SIZE 16
#define KERNEL_SIZE 5 // must be odd
#define STREAM_NUM 10

int cudaEssential(void);
int streamForConcurrency(void);

int main(void)
{
    cudaEssential();
    streamForConcurrency();
}

int cudaEssential(void) 
{
    std::string imagePath = "../Lenna.png";
    std::string outputPath = "./output.png"
    cv::Mat h_src;
    cv::Mat h_dst;
    unsigned char* d_src;
    unsigned char* d_dst;
    CVMethods cvMethod = SOBEL;

    h_src = cv::imread(imagePath, cv::IMREAD_GRAYSCALE);

    int width = src.cols;
    int height = src.rows;
    int step = static_cast<int>(src.step);

    hipMalloc(&d_src, height, width, CV_8UC1);
    hipMalloc(&d_dst, height, width, CV_8UC1);
    // H2D
    hipMemcpy(d_src, src.data, height * step, hipMemcpyHostToDevice);

    dim3 block(CUDA_BLOCK_SIZE,CUDA_BLOCK_SIZE);
    dim3 grid((width + (CUDA_BLOCK_SIZE - 1)) / CUDA_BLOCK_SIZE, (height + (CUDA_BLOCK_SIZE - 1)) / CUDA_BLOCK_SIZE)

    // Kernel
    switch (cvMethod)
    {
    case SOBEL:
        sobelKernel <<<grid, block, 0>>> (d_src, d_dst, width, height, step);
        break;

    case MEAN:
        meanFilterKernel <<<grid, block, 0>>> (d_src, d_dst, width, height, step, KERNEL_SIZE);
        break;

    case MEDIAN:
        medianFilterKernel <<<grid, block, 0>>> (d_src, d_dst, width, height, step, KERNEL_SIZE);
        break;

    default:
        std::cout << "undefined cv method. Shut the program down." << std::endl;
        // memory free
        hipFree(d_src);
        hipFree(d_dst);
        retrun -1;
    }

    // D2H
    hipMemcpy(h_dst.data, d_dst, height * step, hipMemcpyDeviceToHost);

    // save
    cv::imwrite(outputPath, dst);

    // memory free
    hipFree(d_src);
    hipFree(d_dst);

    return 0;
}

int streamForConcurrency(void)
{
    int numImages 50;
    std::string imagePath = "../Lenna.png";
    std::string outputPath = "./output.png";
    // need to something for "*PathVec"
    std::vector<std::string> imagePathVec(numImages);
    std::vector<std::string> outputPathVec(numImages);
    std::vector<cv::Mat> h_srcVec(numImages);
    std::vector<cv::Mat> h_dstVec(numImages);
    std::vector<unsigned char*> d_srcVec(numImages);
    std::vector<unsigned char*> d_dstVec(numImages);
    // for stream management(concurrency)
    std::vector<hipStream_t> streamVec(STREAM_NUM);
    // if you want to manage upload(H2D) or download(D2H)
    // hipStream_t uploadStream;
    // hipStream_t downloadStream;
    
    for (int i = 0; i < numImages; i++){
        h_srcVec[i] = cv::imread(imagePathVec[i], cv::IMREAD_GRAYSCALE);
    }
    
    int width = h_srcVec[0].cols;
    int height = h_srcVec[0].rows;
    int step = static_cast<int>(h_srcVec[0].step);
    
    for (int i = 0; i < numImages; i++) {
        hipMalloc(&d_srcVec[i], height, width, CV_8UC1);
        hipMalloc(&d_dstVec[i], height, width, CV_8UC1);
        // H2D
        hipMemcpy(d_srcVec[i], h_srcVec.data, height * step, hipMemcpyHostToDevice);
    }
    
    dim3 block(CUDA_BLOCK_SIZE,CUDA_BLOCK_SIZE);
    dim3 grid((width + (CUDA_BLOCK_SIZE - 1)) / CUDA_BLOCK_SIZE, (height + (CUDA_BLOCK_SIZE - 1)) / CUDA_BLOCK_SIZE)
    
    // for concurrency
    // Excuted the entire data(images) in units of `subIter`
    int subIterNumByStream = (numImages % STREAM_NUM == 0) ? (numImages / STREAM_NUM) : (numImages / STREAM_NUM + 1);
    int subIterInit = 0;
    int subIterEnd = STREAM_NUM;
    int subStreamIdx = 0;

    for (int i = 0; i < subIterNumByStream; i++) {
        for (int j = subInit; j < subEnd; j++) {
            subStreamIdx = j - (STREAM_NUM * i);
            if (j < numImages) {
                sobelKernel << <grid, block, 0, streamVec[subStreamIdx] >> > (d_srcVec[j], d_dstVec[j], width, height, step);
            }
        }
        subIterInit += STREAM_NUM;
        subIterEnd = (subIterEnd + STREAM_NUM <= numImages) ? (subIterEnd + STREAM_NUM) : numImages;
    }

    for (int i = 0; i < STREAM_NUM; i++) {
        hipStreamDestroy(streamVec[i]);
    }

    for (int i = 0; i < numImages; i++) {
        // D2H
        hipMemcpy(h_dstVec[i].data, d_dstVec[i], height * step, hipMemcpyDeviceToHost);
        
        // save
        cv::imwrite(outputPathVec[i], h_dstVec[i]);
        
        // memory free
        hipFree(d_srcVec[i]);
        hipFree(d_dstVec[i]);
    }

    return 0;
}
